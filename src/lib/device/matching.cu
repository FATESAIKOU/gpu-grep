#include "hip/hip_runtime.h"
extern "C" {

#include "../../cmncateStruct.h"
#include "../../libs.h"
#include "../../commonInc.h"
#include <hip/hip_runtime.h>

MatchRes *matchRecords(MatchOpt *match_opt, MatchBase *match_base);
ResultRec **matching(MatchOpt *match_opt, MatchBase *match_base);
int getScore(MatchOpt *match_opt, int **matching_counter, int word_num);
int ***cpuMatching(MatchOpt *match_opt, MatchBase *match_base);

MatchRes *matchRecords(MatchOpt *match_opt, MatchBase *match_base) {
    MatchRes *match_res = (MatchRes*) malloc(sizeof(MatchRes));

    match_res->result_records = matching(match_opt, match_base);
    match_res->result_length = match_base->record_num;
    // 1. get result_records by calling matching(match_opt, match_base)
    //  (i) match_res->result_records = matching(match_opt, match_base);
    // 2. get result_length from match_base
    //  (i) match_res->result_length = match_base->record_num
    // 3. return match_res

    return match_res;
}

ResultRec **matching(MatchOpt *match_opt, MatchBase *match_base) {
    ResultRec **result_records = (ResultRec**) malloc(sizeof(ResultRec*) * (match_base->record_num));

    int ***matching_counters = cpuMatching(match_opt, match_base);
    int i;
    for (i = 0; i < match_base->record_num; i ++) {
        result_records[i] = (ResultRec*) malloc(sizeof(ResultRec));
        result_records[i]->score = getScore(match_opt, matching_counters[i], match_base->record_num);
        result_records[i]->record = match_base->sample_records[i];
    }

    // 1. get matching_counter by calling device_ matching. Ps matching_counter[record_id][attribute_id][word_id] = appear_time.
    //  (i)  matching_counters = gpuMatching<<<Block, Thread>>>(match_opt, match_base)  ::need hipMemcpy(in & out)
    // 2. using matching_counter above, generate structure ResultRec
    //  for i in 0..match_base->record_num
    //  (i)  result_records[record_id]->score = getScore(match_opt, matching_counters[record_id], record_num)
    //  (ii) result_records[record_id]->record = match_base->sample_records[record_id];
    // 3. return result_records

    return result_records;
}

int getScore(MatchOpt *match_opt, int **matching_counter, int word_num) {
    int score = 0;

    // 1. count out score
    int i, j;
    for (i = 0; i < COLUMN_NUM; i ++) {
        for (j = 0; j < word_num; j ++) {
            score += matching_counter[i][j];
        }
    }

    // 2. return the score

    return score;
}

int ***cpuMatching(MatchOpt *match_opt, MatchBase *match_base) {
    char *aim_word;
    int rec_id, attr_id, word_id;
    int *scores = (int*) malloc(sizeof(int) * (match_base->record_num));

    int ***match_counter = (int***) malloc(sizeof(int**) * match_base->record_num);
    for (rec_id = 0; rec_id < match_base->record_num; rec_id ++) {

        match_counter[rec_id] = (int**) malloc(sizeof(int*) * COLUMN_NUM);
        for (attr_id = 0; attr_id < COLUMN_NUM; attr_id ++) {

            match_counter[rec_id][attr_id] = (int*) malloc(sizeof(int) * match_opt->match_str_num);
            for (word_id = 0; word_id < match_opt->match_str_num; word_id ++) {
                aim_word = match_opt->match_strs[word_id];

                if (strstr(match_base->sample_records[rec_id][attr_id], aim_word) != NULL) {
                    match_counter[rec_id][attr_id][word_id] = 1;
                } else {
                    match_counter[rec_id][attr_id][word_id] = 0;
                }
            }
        }
    }

    return match_counter;
}

}

__global__ void gpuMatching(MatchOpt *match_opt, MatchBase *match) {
    // ThreadIdx == 0
    // 1. wait for all thread completing it's task
    // 2. merge all of the matching counter
    // 3. complete match_counter
    //
    // ThreadIdx == 1 - warpsize
    // 1. malloc a memory space to store the length is warpsize - 1 records' matching counter
    // 2. matching & record
    // 3. wait for somebody to merge all of these matching counter
}
