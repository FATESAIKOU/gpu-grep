#include "hip/hip_runtime.h"
extern "C" {

#include "../../cmncateStruct.h"
#include "../../libs.h"
#include <hip/hip_runtime.h>

MatchRes *matchRecords(MatchOpt *match_opt, MatchBase *match_base);
ResultRec **matching(MatchOpt *match_opt, MatchBase *match_base);
int getScore(MatchOpt *match_opt, int **matching_counter, int word_num);

MatchRes *matchRecords(MatchOpt *match_opt, MatchBase *match_base) {
    MatchRes *match_res = (MatchRes*) malloc(sizeof(MatchRes));

    // 1. get result_record by calling matching(match_opt, match_base)
    //  (i) match_res->result_record = matching(match_opt, match_base);
    // 2. get result_length from match_base
    //  (i) match_res->result_length = match_base->record_num
    // 3. return match_res

    return match_res;
}

ResultRec **matching(MatchOpt *match_opt, MatchBase *match_base) {
    ResultRec **result_records = (ResultRec**) malloc(sizeof(ResultRec*) * (match_base->record_num));

    // 1. get matching_counter by calling device_ matching. Ps matching_counter[record_id][attribute_id][word_id] = appear_time.
    //  (i)  matching_counters = gpuMatching<<<Block, Thread>>>(match_opt, match_base)  ::need hipMemcpy(in & out)
    // 2. using matching_counter above, generate structure ResultRec
    //  for i in 0..match_base->record_num
    //  (i)  result_records[record_id]->score = getScore(match_opt, match_counters[record_id], record_num)
    //  (ii) result_records[record_id]->record = match_base->sample_records[record_id];
    // 3. return result_records

    return result_records;
}

int getScore(MatchOpt *match_opt, int **matching_counter, int word_num) {
    int score = 0;

    // 1. count out score
    // 2. return the score

    return score;
}

}

__global__ void gpuMatching(MatchOpt *match_opt, MatchBase *match) {
    // ThreadIdx == 0
    // 1. wait for all thread completing it's task
    // 2. merge all of the matching counter
    // 3. complete match_counter
    //
    // ThreadIdx == 1 - warpsize
    // 1. malloc a memory space to store the length is warpsize - 1 records' matching counter
    // 2. matching & record
    // 3. wait for somebody to merge all of these matching counter
}
