#include "hip/hip_runtime.h"
extern "C" {

#include "../../cmncateStruct.h"
#include "../../libs.h"
#include "../../commonInc.h"
#include <hip/hip_runtime.h>

#define ThreadNum 1024
#define BlockNum 32

MatchRes *genMatchRes(MatchOpt *match_opt, MatchBase *match_base);
RecordInfo *matching(MatchOpt *match_opt, MatchBase *match_base);
char *loadFile(MatchOpt* match_opt, MatchBase *match_base);
RecordInfo *loadRecord(MatchBase *match_base);
void getRecordScore(MatchOpt *match_opt, char *datas, RecordInfo *record_info, int rec_num);
char **getMatchStrs(MatchOpt *match_opt);
__global__ void gpuMatching(char *datas, RecordInfo *rec_info, char **match_strs, int str_num, int rec_num, int attr_num, int base);
__device__ int d_strstr(const char *a, const char *b);
__device__ int d_strlen(const char *s);


MatchRes *genMatchRes(MatchOpt *match_opt, MatchBase *match_base) {
    MatchRes *match_result = (MatchRes*) malloc(sizeof(MatchRes));

    hipDeviceReset();
    match_result->record_info = matching(match_opt, match_base);
    match_result->rec_entry = match_base->rec_entry;

    return match_result;
}

RecordInfo *matching(MatchOpt *match_opt, MatchBase *match_base) {
    // 1. load matching base to Device
    RecordInfo *device_rec_info = loadRecord(match_base);
    char *device_datas = loadFile(match_opt, match_base);
    int rec_num = match_base->rec_entry;

    // 2. get each record score
    getRecordScore(match_opt, device_datas, device_rec_info, rec_num);
    hipMemcpy(match_base->record_info, device_rec_info, sizeof(RecordInfo) * match_base->rec_entry, hipMemcpyDeviceToHost);

    // 3. free cuda memory
    hipFree(device_rec_info);

    // 4. return RecordInfo
    return match_base->record_info;
}

char *loadFile(MatchOpt* match_opt, MatchBase *match_base) {
    char *device_datas;
    char *host_datas = match_base->datas;
    size_t data_size = sizeof(char) * match_base->data_len;

    hipMalloc(&device_datas, data_size);
    hipMemcpy(device_datas, host_datas, data_size, hipMemcpyHostToDevice);

    return device_datas;
}

RecordInfo *loadRecord(MatchBase *match_base) {
    RecordInfo *host_rec_info = match_base->record_info;
    long unsigned int rec_info_size = sizeof(RecordInfo) * match_base->rec_entry;

    RecordInfo *device_rec_info;
    hipMalloc(&device_rec_info, rec_info_size);
    hipMemcpy(device_rec_info, host_rec_info, rec_info_size, hipMemcpyHostToDevice);

    return device_rec_info;
}

void getRecordScore(MatchOpt *match_opt, char *datas, RecordInfo *record_info, int rec_num) {
    // matching
    char **match_strs = getMatchStrs(match_opt);
    int match_str_num = match_opt->match_str_num;
    int base, round_limit = ceil(rec_num / (float) (BlockNum * ThreadNum));
    int i;

    for (i = 0; i < round_limit; i ++) {
        base = i * BlockNum * ThreadNum;
        gpuMatching<<<BlockNum, ThreadNum>>>(datas, record_info, match_strs, match_str_num, rec_num, COLUMN_NUM, base);
    }
}

char **getMatchStrs(MatchOpt *match_opt) {
    int match_str_num = match_opt->match_str_num;
    char **device_match_strs;
    hipMallocManaged(&device_match_strs, sizeof(char*) * match_str_num);

    int i, str_len;
    char *device_str;
    for (i = 0; i < match_str_num; i ++) {
        str_len = strlen(match_opt->match_strs[i]) + 1;
        hipMalloc(&device_str, sizeof(char) * str_len);
        hipMemcpy(device_str, match_opt->match_strs[i], sizeof(char) * str_len, hipMemcpyHostToDevice);

        device_match_strs[i] = device_str;
    }

    return device_match_strs;
}

}

__global__ void gpuMatching(char *datas, RecordInfo *rec_info, char **match_strs, int str_num, int rec_num, int attr_num, int base) {
    int index = base + blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= rec_num) return;

    int i, j = 0;
    int offset;

    RecordInfo *aim = &rec_info[index];
    int score = 0;
    for (i = 0; i < attr_num; i ++) {
        for (j = 0; j < str_num; j ++) {
            offset = aim->attr_offset[i];
            while ((offset = d_strstr(datas + offset, match_strs[j])) >= 0) {
                score = score + 1;
            }
        }
    }

    aim->score = score;
}

__device__ int d_strstr(const char *a, const char *b) {
    int i, j;
    int a_len = d_strlen(a);
    int b_len = d_strlen(b);
    int loop_limit = a_len - b_len + 1;


    for (i = 0; i < loop_limit; i ++) {
        for (j = 0; j < b_len && a[i + j] == b[j]; j ++) ;

        if (j == b_len) return i;
    }

    return -1;
}

__device__ int d_strlen(const char *s) {
    int i = 0;
    while (s[i] != '\0') i ++;

    return i;
}
