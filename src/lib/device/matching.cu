#include "hip/hip_runtime.h"
extern "C" {

#include "../../cmncateStruct.h"
#include "../../libs.h"
#include "../../commonInc.h"
#include <hip/hip_runtime.h>

#define ThreadNum 1024
#define BlockNum 32

MatchRes *genMatchRes(MatchOpt *match_opt, MatchBase *match_base);
RecordInfo *matching(MatchOpt *match_opt, MatchBase *match_base);
char *loadFile(MatchOpt* match_opt, MatchBase *match_base);
RecordInfo *loadRecord(MatchBase *match_base);
int *getRecordScore(MatchOpt *match_opt, char *datas, RecordInfo *record_info, int rec_num);
char **getMatchStrs(MatchOpt *match_opt);
__global__ void gpuMatching(char *datas, RecordInfo *rec_info, char **match_strs, int str_num, int rec_num, int attr_num, int base);
__device__ int d_strstr(char *a, char *b);
__device__ int d_strlen(char *s);


MatchRes *genMatchRes(MatchOpt *match_opt, MatchBase *match_base) {
    MatchRes *match_result = (MatchRes*) malloc(sizeof(MatchRes));

    match_result->record_info = matching(match_opt, match_base);
    match_result->rec_entry = match_base->rec_entry;

    return match_result;
}

RecordInfo *matching(MatchOpt *match_opt, MatchBase *match_base) {
    // 1. load matching base to Device
    RecordInfo *device_rec_info = loadRecord(match_base);
    char *device_datas = loadFile(match_opt, match_base);
    int rec_num = match_base->rec_entry;

    // 2. get each attr score
    getRecordScore(match_opt, device_datas, device_rec_info, rec_num);
    hipDeviceSynchronize();

    // 4. return RecordInfo
    return device_rec_info;
}

char *loadFile(MatchOpt* match_opt, MatchBase *match_base) {
    char *device_datas;
    char *host_datas = match_base->datas;
    size_t data_size = sizeof(char) * match_base->data_len;

    hipMalloc(&device_datas, data_size);
    hipMemcpy(device_datas, host_datas, data_size, hipMemcpyHostToDevice);

    return device_datas;
}

RecordInfo *loadRecord(MatchBase *match_base) {
    RecordInfo *host_rec_info = match_base->record_info;
    int rec_info_size = sizeof(RecordInfo) * match_base->rec_entry;

    RecordInfo *device_rec_info;
    hipMallocManaged(&device_rec_info, rec_info_size);
    memcpy(device_rec_info, host_rec_info, rec_info_size);


    return device_rec_info;
}

int *getRecordScore(MatchOpt *match_opt, char *datas, RecordInfo *record_info, int rec_num) {
    // matching
    char **match_strs = getMatchStrs(match_opt);
    int match_str_num = match_opt->match_str_num;
    int base, round_limit = ceil(rec_num / (float) (BlockNum * ThreadNum));
    int i;
    for (i = 0; i < round_limit; i ++) {
        base = i * BlockNum * ThreadNum;
        gpuMatching<<<BlockNum, ThreadNum>>>(datas, record_info, match_strs, match_str_num, rec_num, COLUMN_NUM, base);
    }

    return NULL;
}

char **getMatchStrs(MatchOpt *match_opt) {
    int match_str_num = match_opt->match_str_num;
    char **device_match_strs;
    hipMallocManaged(&device_match_strs, sizeof(char*) * match_str_num);

    int i, str_len;
    char *device_str;
    for (i = 0; i < match_str_num; i ++) {
        str_len = strlen(match_opt->match_strs[i]);
        hipMalloc(&device_str, sizeof(char) * str_len);
        hipMemcpy(device_str, match_opt->match_strs[i], sizeof(char) * str_len, hipMemcpyHostToDevice);

        device_match_strs[i] = device_str;
    }

    return device_match_strs;
}

}

__global__ void gpuMatching(char *datas, RecordInfo *rec_info, char **match_strs, int str_num, int rec_num, int attr_num, int base) {
    int index = base + blockIdx.x * blockDim.x + threadIdx.x;

    int i, j = 0;
    int offset;

    RecordInfo *aim = rec_info + index;
    for (i = 0; i < attr_num; i ++) {
        for (j = 0; j < str_num; j ++) {
            offset = rec_info[index].attr_offset[i];
            while  ((offset = d_strstr(datas + offset, match_strs[j])) > 0) {
                aim->score ++;
            }
        }
    }
}

__device__ int d_strstr(char *a, char *b) {
    int i, j;
    int a_len = d_strlen(a);
    int b_len = d_strlen(b);
    int loop_limit = a_len - b_len + 1;

    for (i = 0; i < loop_limit; i ++) {
        for (j = 0; j < b_len && a[i + j] == b[j]; j ++) ;

        if (j == b_len) return i;
    }

    return NULL;
}

__device__ int d_strlen(char *s) {
    int i = 0;
    while (s[i] != '\0') i ++;

    return i;
}
