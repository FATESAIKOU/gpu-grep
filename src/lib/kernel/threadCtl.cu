extern "C" {

#include "../../cmncateStruct.h"
#include "../../commonInc.h"
#include <hip/hip_runtime.h>

MatchOpt *genMatchOpt(ThreadOpt *thread_opt);
MatchBase *genMatchBase(ThreadOpt *thread_opt, char *filename);
RecordInfo *readRecordsInfo(char *file_content, char *begin, int *entry);
char *readFile(char *filename, size_t *file_len);
size_t getFileSize(char *filename);
int getAttrId(char *str);
void checkBase(MatchBase *match_base, int attr_num);


MatchOpt *genMatchOpt(ThreadOpt *thread_opt) {
    MatchOpt *match_opt = (MatchOpt*) malloc(sizeof(MatchOpt));

    match_opt->match_strs = thread_opt->match_strs;
    match_opt->match_mode = thread_opt->match_mode;
    match_opt->error_rate = thread_opt->error_rate;
    match_opt->match_str_num = thread_opt->match_str_num;
    match_opt->insensitive = thread_opt->insensitive;

    match_opt->match_relations = thread_opt->match_relations;

    return match_opt;
}

MatchBase *genMatchBase(ThreadOpt *thread_opt, char *filename) {
    MatchBase *match_base = (MatchBase*) malloc(sizeof(MatchBase));
    size_t file_len;
    char *file_content = readFile(filename, &file_len);
    int record_num;

    match_base->datas = file_content;
    match_base->record_info = readRecordsInfo(file_content, thread_opt->begin_at, &record_num);
    match_base->rec_entry = record_num;
    match_base->data_len = file_len;

    return match_base;
}

RecordInfo *readRecordsInfo(char *file_content, char *begin, int *entry) {
    int limit = 1024;
    int begin_id = getAttrId(begin);

    // initialize all of the record_info
    RecordInfo *records_info = (RecordInfo*) malloc(sizeof(RecordInfo) * limit);
    memset(records_info, 0, sizeof(RecordInfo) * limit);

    char *attr = strtok(file_content, "\n");
    int record_id = 0;
    int attr_id;
    while(attr != NULL) {
        attr_id = getAttrId(attr);
        if (attr_id == begin_id) {
            record_id ++;
            if (record_id == limit) {
                limit += 1024;
                records_info = (RecordInfo*) realloc(records_info, sizeof(RecordInfo) * limit);
            }
            memset(records_info + record_id, 0, sizeof(RecordInfo));
            records_info[record_id].record_offset = attr - file_content;
        } else if (attr_id != -1) {
            records_info[record_id].attr_offset[attr_id] = attr - file_content;
        }

        attr = strtok(NULL, "\n");
    }

    records_info = (RecordInfo*) realloc(records_info, sizeof(RecordInfo) * record_id);
    *entry = record_id + 1;

    return records_info;
}

char *readFile(char *filename, size_t *file_len) {
    int fd = open(filename, O_RDONLY, 0644);
    *file_len = getFileSize(filename) + 1;

    char *file_content = (char*) mmap(NULL, *file_len, PROT_READ | PROT_WRITE, MAP_PRIVATE, fd, 0);
    file_content[*file_len] = '\0';

    return file_content;
}

size_t getFileSize(char *filename) {
    struct stat st;
    stat(filename, &st);

    return (st.st_size);
}

int getAttrId(char *str) {
    if (str[0] != '@') return -1;

    switch (str[1]) {
        case 'i': return 0;
        case 't': return 1;
        case 'p': return 2;
        case 'c': return 3;
        case 'd': return 4;
        case 'f': return 5;
        case '$': return 6;
        case 'a': return 7;
        case 'k': return 8;
        case '_': return 9;
    }

    return -1;
}

void checkBase(MatchBase *match_base, int attr_num) {
    char *base = match_base->datas;
    RecordInfo *rec_info = match_base->record_info;
    int rec_num = match_base->rec_entry;
    int i, j;

    int offset;
    for (i = 0; i < rec_num; i ++) {
        printf("Record %d---\n", i);
        for (j = 0; j < attr_num; j ++) {
            offset = rec_info[i].attr_offset[j];
            if (offset != 0) {
                printf("%s\n", base + offset);
            }
        }
    }
}

}
